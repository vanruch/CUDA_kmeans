#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include "hip/device_functions.h"


#include <hip/hip_runtime_api.h>

#include <>
#include <iostream>
#include <fstream>
#include <stdio.h>
#include <string>
#include <numeric>
#include <ctime>

#include <thrust\host_vector.h>
#include <thrust\device_vector.h>
#include <thrust\sort.h>
#include <thrust\copy.h>
#include <thrust\device_ptr.h>
#include <thrust\device_malloc.h>
#include <thrust\for_each.h>
#include <thrust\transform.h>
#include <thrust/execution_policy.h>
#include <thrust\scan.h>

#define SIZE 50000
#define K 3
#define LO -1000.0
#define HI 1000.0

using namespace std;
using namespace thrust;

void readData(host_vector<float3> &vec) {
	
	for (int i = 0; i < SIZE; i++)
	{
		float3 f;
		f.x= LO + static_cast <float> (rand()) / (static_cast <float> (RAND_MAX / (HI - LO)));
		f.y= LO + static_cast <float> (rand()) / (static_cast <float> (RAND_MAX / (HI - LO)));
		f.z= LO + static_cast <float> (rand()) / (static_cast <float> (RAND_MAX / (HI - LO)));
		vec.push_back(f);
	}

}

struct dist_functor {
	float3* centr;
	
	dist_functor(float3* _centr) : centr(_centr) {}

	__host__ __device__ int operator()(const float3 point) {
		float dist = INFINITY;
		int r = 0;
		for (int i = 0; i < K; i++)
		{
			int d = (centr[i].x - point.x)*(centr[i].x - point.x)+
				(centr[i].y - point.y)*(centr[i].y - point.y)+
				(centr[i].z - point.z)*(centr[i].z - point.z);
			if (d < dist) {
				r = i;
				dist = d;
			}
		}
		return r;
	}
};

struct sum_functor {
	float3* centr;
	int* sums;

	sum_functor(float3* _centr, int* _sums) : centr(_centr), sums(_sums) {}

	 __device__ void operator()(const thrust::tuple<float3, int> &point) {
		int i = point.get<1>();
		atomicAdd(&centr[i].x, point.get<0>().x);
		atomicAdd(&centr[i].y, point.get<0>().y);
		atomicAdd(&centr[i].z, point.get<0>().z);
		atomicAdd(&sums[i], 1);
	}
};

struct mean_functor
{
	mean_functor()
	{

	}
	__host__ __device__ float3 operator()(float3 p, int c) {
		float3 t;
		t.x = p.x / (float)c;
		t.y = p.y / (float)c;
		t.z = p.z / (float)c;
		return t;
	}

};

struct eq_functor {

	__host__ __device__ bool operator()(const thrust::tuple<int,int> &t) {
		return  t.get<1>() != t.get<0>();
	}
};

host_vector<int> cudaKMeans(host_vector<float3> &points) {
	device_vector<float3> d_points = points;
	host_vector<float3> centroids;
	for (int i = 0; i < K; i++)
	{
		centroids.push_back(points[i]);
	}
	device_vector<float3> d_centroids = centroids;
	device_vector<int> d_indexes(points.size(), -1);
	device_vector<int> d_new_indexes(points.size());
	device_vector<int> cnt(K);
	int diff=1;
	while (diff>0)
	{
		thrust::transform(d_points.begin(), d_points.end(), d_new_indexes.begin(), dist_functor(raw_pointer_cast(d_centroids.data())));
		diff = thrust::count_if(make_zip_iterator(thrust::make_tuple(d_indexes.begin(), d_new_indexes.begin())),
			make_zip_iterator(thrust::make_tuple(d_indexes.end(), d_new_indexes.end())), eq_functor());		
		thrust::copy(d_new_indexes.begin(), d_new_indexes.end(), d_indexes.begin());
		thrust::fill(cnt.begin(), cnt.end(), 0);
		thrust::fill(d_centroids.begin(), d_centroids.end(), float3());
		thrust::for_each(make_zip_iterator(thrust::make_tuple(d_points.begin(), d_indexes.begin())),
			make_zip_iterator(thrust::make_tuple(d_points.end(), d_indexes.end())),
			sum_functor(raw_pointer_cast(d_centroids.data()), raw_pointer_cast(cnt.data())));
		thrust::transform(d_centroids.begin(), d_centroids.end(), cnt.begin(), d_centroids.begin(), mean_functor());
	}
	return host_vector<int>(d_indexes);
}

//--------------HOST------------------
void new_ind(host_vector<float3> &centr, host_vector<float3> &points, host_vector<int> &indexes) {
	for (int j = 0; j < SIZE; j++)
	{
		float dist = INFINITY;

		for (int i = 0; i < K; i++)
		{
			int d = (centr[i].x - points[j].x)*(centr[i].x - points[j].x) +
				(centr[i].y - points[j].y)*(centr[i].y - points[j].y) +
				(centr[i].z - points[j].z)*(centr[i].z - points[j].z);
			if (d < dist) {
				indexes[j] = i;
				dist = d;
			}
		}
	}
}

host_vector<int> hostKMeans(host_vector<float3> &points) {
	host_vector<float3> centroids;
	for (int i = 0; i < K; i++)
	{
		centroids.push_back(points[i]);
	}
	host_vector<int> indexes(points.size(), -1);
	host_vector<int> new_indexes(points.size());
	int diff = 1;
	while (diff > 0) {
		new_ind(centroids, points, new_indexes);
		diff = 0;
		for (int i = 0; i < SIZE; i++)
		{
			if (new_indexes[i] != indexes[i]) {
				indexes[i] = new_indexes[i];
				diff++;
			}
		}
		host_vector<int> cnt(K);
		for (size_t i = 0; i < K; i++)
		{
			centroids[i] = float3();
		}
		for (int i = 0; i < SIZE; i++)
		{
			cnt[indexes[i]]++;
			centroids[indexes[i]].x += points[i].x;
			centroids[indexes[i]].y += points[i].y;
			centroids[indexes[i]].z += points[i].z;
		}
		for (size_t i = 0; i < K; i++)
		{
			centroids[i].x /= cnt[i];
			centroids[i].y /= cnt[i];
			centroids[i].z /= cnt[i];
		}
	}
	return indexes;
}


int main()
{


	
	host_vector<float3> points;



	cout << "gen\n";
	readData(points);
	cout << "calc 1\n";
	clock_t begin = clock();
	auto v = cudaKMeans(points);
	clock_t end = clock();
	cout << "CUDA: " << double(end - begin) / CLOCKS_PER_SEC << endl;
	
	begin = clock();
	auto v2 = hostKMeans(points);
	end = clock();
	cout << "HOST: " << double(end - begin) / CLOCKS_PER_SEC << endl;

	for (int i = 0; i < SIZE; i++)
	{
		if (v[i] != v2[i]) {
			cout << "ERR";
			return 0;
		}
	}
	int a[K] = { 0 };
	ofstream myfile;
	myfile.open("kmeans.csv");
	myfile << "X,Y,Z,cluster\n";
	

	for (int i = 0; i < SIZE; i++)
	{

		a[v[i]]++;
		myfile << points[i].x << ',' << points[i].y << ',' << points[i].z << ',' << v[i] << endl;
	}
	myfile.close();
	for (size_t i = 0; i < K; i++)
	{
		cout << a[i] << endl;
	}
	return 0;
}